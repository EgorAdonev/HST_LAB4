#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <fcntl.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define MEGA 1024 * 1024
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort=true){
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
 
__global__ void multiplyMatrix(int* matrix, int* res_matrix, int size){
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int column = blockDim.y * blockIdx.y + threadIdx.y;
    if (row < size && column < size){
        int sum = 0;
        for (int rank = 0; rank < size; rank++)
            sum += matrix[row * size + rank] * matrix[rank * size + column];
        res_matrix[row * size + column] = sum;
    }
}

int main(int argc, char *argv[]){
    float start_time = (float)clock();
    int size_of_data = 0;
    FILE* fp = fopen("input", "rb");
    int count_of_matrices;
    fread(&count_of_matrices, sizeof(int), 1, fp);
    int** matrices = (int**) malloc(count_of_matrices * sizeof(int*));
    int* ranks_of_matrices = (int*) malloc(count_of_matrices * sizeof(int));
    for (int index_of_matrix = 0; index_of_matrix < count_of_matrices; index_of_matrix++){
        int rang;
        fread(&rang, sizeof(int), 1, fp);
        ranks_of_matrices[index_of_matrix] = rang;
        matrices[index_of_matrix] = (int*) malloc(rang * rang * sizeof(int));
        for (int row = 0; row < rang; row++)
            for (int column = 0; column < rang; column++)
                fread(&matrices[index_of_matrix][row * rang + column], sizeof(int), 1, fp);
        size_of_data += rang * rang * sizeof(int);
    }
    fclose(fp);
    for (int index_of_matrix = 0; index_of_matrix < count_of_matrices; index_of_matrix++){
        int* buffer_matrix = (int*) malloc(ranks_of_matrices[index_of_matrix] * ranks_of_matrices[index_of_matrix] * sizeof(int));
        int* cuda_matrix;
        gpuErrchk( hipMalloc((void**)&cuda_matrix, ranks_of_matrices[index_of_matrix] * ranks_of_matrices[index_of_matrix] * sizeof(int)) );
        int* cuda_res_matrix;
        gpuErrchk( hipMalloc((void**)&cuda_res_matrix, ranks_of_matrices[index_of_matrix] * ranks_of_matrices[index_of_matrix] * sizeof(int)) );
        gpuErrchk( hipMemcpy(cuda_matrix, matrices[index_of_matrix], ranks_of_matrices[index_of_matrix] * ranks_of_matrices[index_of_matrix] * sizeof(int), hipMemcpyHostToDevice    ) );
        dim3 threadsPerBlock(ranks_of_matrices[index_of_matrix], ranks_of_matrices[index_of_matrix]);
        dim3 numBlocks(ranks_of_matrices[index_of_matrix] / threadsPerBlock.x, ranks_of_matrices[index_of_matrix] / threadsPerBlock.y);
        multiplyMatrix<<<numBlocks, threadsPerBlock>>>(cuda_matrix, cuda_res_matrix, ranks_of_matrices[index_of_matrix]);
        gpuErrchk( hipMemcpy(buffer_matrix, cuda_res_matrix, ranks_of_matrices[index_of_matrix] * ranks_of_matrices[index_of_matrix] * sizeof(int), hipMemcpyDeviceToHost));
        for (int row = 0; row < ranks_of_matrices[index_of_matrix]; row++)
            for (int column = 0; column < ranks_of_matrices[index_of_matrix]; column++)
                matrices[index_of_matrix][row * ranks_of_matrices[index_of_matrix] + column] = buffer_matrix[row * ranks_of_matrices[index_of_matrix] + column];
        free(buffer_matrix);
        gpuErrchk( hipFree(cuda_matrix) );
        gpuErrchk( hipFree(cuda_res_matrix) );
    }
    fp = fopen("output.txt", "w");
    fprintf(fp, "%d\n", count_of_matrices);
    for (int index_of_matrix = 0; index_of_matrix < count_of_matrices; index_of_matrix++){
        fprintf(fp, "%d\n", ranks_of_matrices[index_of_matrix]);
        for (int row = 0; row < ranks_of_matrices[index_of_matrix]; row++){
            for (int column = 0; column < ranks_of_matrices[index_of_matrix]; column++)
                fprintf(fp, "%d ", matrices[index_of_matrix][row * ranks_of_matrices[index_of_matrix] + column]);
            fputs("\n", fp);
        }
    }
    for (int index_of_matrix = 0; index_of_matrix < count_of_matrices; index_of_matrix++){
        free(matrices[index_of_matrix]);
    }
    free(matrices);
    free(ranks_of_matrices);
    float end_time = ((float)clock()) - start_time;
    fprintf(fp, "Count time: %f s\n", (float)size_of_data / (MEGA));
    fprintf(fp, "Size of data: %f Mb\n", end_time / 1000000);
    printf("Success!\n");
    return 0;
}
